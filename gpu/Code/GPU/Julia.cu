#include "hip/hip_runtime.h"
// Inclusion de la bibliothèque pour la sauvegarde d'images
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

// Structure pour représenter les nombres complexes
struct hipComplex {
    float r; // Partie réelle
    float i; // Partie imaginaire

    // Constructeur pour initialiser un nombre complexe avec des valeurs données
    __device__ hipComplex(float a, float b) : r(a), i(b) {}

    // Méthode pour calculer le carré de la magnitude d'un nombre complexe
    __device__ float magnitude2(void) { return r * r + i * i; }

    // Surcharge de l'opérateur de multiplication pour les nombres complexes
    __device__ hipComplex operator*(const hipComplex &a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }

    // Surcharge de l'opérateur d'addition pour les nombres complexes
    __device__ hipComplex operator+(const hipComplex &a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

// Kernel CUDA pour calculer la fractale de Julia
__global__ void juliaKernel(unsigned char *ptr, int width, int height) {
    // Calcul des coordonnées du pixel dans l'image
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * width;

    // Définition de la constante de Julia et du facteur d'échelle
    const float scale = 1.5;
    float jx = scale * (float)(width / 2 - x) / (width / 2);
    float jy = scale * (float)(height / 2 - y) / (height / 2);
    hipComplex c(-0.8, 0.156); // Constante de Julia (peut être modifiée pour changer la forme de la fractale)
    hipComplex a(jx, jy); // Nombre complexe correspondant aux coordonnées du pixel

    // Calcul de la fractale de Julia
    int iterations = 0;
    while (iterations < 200 && a.magnitude2() < 1000) {
        a = a * a + c;
        iterations++;
    }

    // Attribution des valeurs de couleurs au pixel
    ptr[offset * 4 + 0] = iterations * iterations % 256; // Rouge
    ptr[offset * 4 + 1] = iterations % 256;               // Vert
    ptr[offset * 4 + 2] = iterations % 256;               // Bleu
    ptr[offset * 4 + 3] = 255;                            // Alpha
}

int main() {
    int DIM; // Résolution de l'image

    // Demande à l'utilisateur la résolution de l'image
    std::cout << "Veuillez saisir la valeur de la résolution de l'image : ";
    std::cin >> DIM;

    // Allocation de mémoire pour l'image sur le GPU
    unsigned char *dev_bitmap;
    unsigned char *bitmap = new unsigned char[4 * DIM * DIM];

    hipMalloc((void **)&dev_bitmap, sizeof(unsigned char) * 4 * DIM * DIM);

    // Définition de la taille des blocs et des grilles pour le kernel CUDA
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(DIM / threadsPerBlock.x, DIM / threadsPerBlock.y);

    // Création d'événements CUDA pour mesurer le temps d'exécution du kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Appel du kernel CUDA pour calculer la fractale de Julia
    juliaKernel<<<numBlocks, threadsPerBlock>>>(dev_bitmap, DIM, DIM);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calcul du temps d'exécution du kernel
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Affichage du temps d'exécution du kernel
    std::cout << "Temps d'exécution du kernel : " << milliseconds << " ms" << std::endl;

    // Copie des données de l'image du GPU vers le CPU
    hipMemcpy(bitmap, dev_bitmap, sizeof(unsigned char) * 4 * DIM * DIM, hipMemcpyDeviceToHost);

    // Libération de la mémoire allouée sur le GPU
    hipFree(dev_bitmap);

    // Sauvegarde de l'image fractale au format PNG
    auto start_time = std::chrono::high_resolution_clock::now();
    stbi_write_png("julia_cuda.png", DIM, DIM, 4, bitmap, DIM * 4);
    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

    // Affichage de la confirmation de sauvegarde et du temps total d'exécution du programme
    std::cout << "Image sauvegardée !" << std::endl;
    std::cout << "Temps d'exécution : " << duration << " ms" << std::endl;
    std::cout << "Temps d'exécution en minutes : " << duration / 60000 << " min " << (duration % 60000) / 1000 << " s" << std::endl;

    // Libération de la mémoire allouée pour l'image sur le CPU
    delete[] bitmap;

    return 0;
}

//ESIEA - 4A - Thomas COSSET - Jean-Juc LAURENT - Théo BACHELERY